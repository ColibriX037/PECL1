#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

using namespace std;

void showMatriz(int matriz[], int anchura);
int * generateMatriz();
void generateSeeds(int matriz[],int cantidad);
bool checkFull(int matriz[], int tamano);

hipError_t cudaStatus;
bool partida_enCurso = true;

__global__ void mov_upK(int *matriz, int anchura, int altura) {

	int x = threadIdx.x;

	int *vector = (int*)malloc(sizeof(int)*anchura*altura);
	for (int i = 0; i < anchura*altura; i++)
	{
		vector[i] = 0;
	}

	int *aux = (int*)malloc(sizeof(int)*anchura*altura);
	for (int i = 0; i < anchura*altura; i++)
	{
		aux[i] = 0;
	}


	int posicion_Vector = 0;
	for (int i = 0; i < altura; i++)
	{
		if (matriz[i*anchura + x] != 0) {
			vector[posicion_Vector] = matriz[i*anchura + x];
			posicion_Vector++;
		}
	}

	int posicion_aux = 0;
	for (int j = 0; j < altura; j++)
	{
		if (vector[j] == vector[j+1])
		{
			aux[posicion_aux] = vector[j] * 2;
			j++;
		}
		else
		{
			aux[posicion_aux] = vector[j];
		}
		
		posicion_aux++;
	}
	
	for (int k = 0; k < altura; k++)
	{
		if (!aux[k])
		{
			aux[k] = 0;
		}
	}

	for (int i = 0; i < altura; i++)
	{
		matriz[i*anchura + x] = aux[i];
	}
}


hipError_t move_up(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto *sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	mov_upK <<< 1,ancho  >>> (dev_m,ancho,alto);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto *sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__global__ void mov_downK(int *matriz, int anchura, int altura) {
	int x = threadIdx.x;

	int *vector = (int*)malloc(sizeof(int)*anchura*altura);
	for (int i = 0; i < anchura*altura; i++)
	{
		vector[i] = 0;
	}

	int *aux = (int*)malloc(sizeof(int)*anchura*altura);
	for (int i = 0; i < anchura*altura; i++)
	{
		aux[i] = 0;
	}

	int posicion_Vector = 0;
	for (int i = altura - 1; i >= 0; i--)
	{
		if (matriz[i*anchura + x] != 0) {
			vector[posicion_Vector] = matriz[i*anchura + x];
			posicion_Vector++;
		}
	}

	int posicion_aux = 0;
	for (int j = 0; j < altura; j++)
	{
		if (vector[j] == vector[j + 1])
		{
			aux[posicion_aux] = vector[j] * 2;
			j++;
		}
		else
		{
			aux[posicion_aux] = vector[j];
		}
		posicion_aux++;
	}

	for (int k = 0; k < altura; k++)
	{
		if (!aux[k])
		{
			aux[k] = 0;
		}
	}
			
	for (int i = 0; i < altura; i++)
	{
		matriz[(altura-1-i)*anchura + x] = aux[i];
	}
}

hipError_t move_down(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	mov_downK << < 1, ancho >> > (dev_m, ancho, alto);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__global__ void mov_leftK(int * matriz[]) {

}

hipError_t move_left(int * matriz[]) {
	hipError_t cudaStatus;
	return cudaStatus;
}

__global__ void mov_rightK(int * matriz[]) {

}

hipError_t move_right(int * matriz[]) {
	hipError_t cudaStatus;
	return cudaStatus;
}


int main()
{
	hipError_t cudaStatus;
	srand(time(NULL));

	int ancho = 4;
	int alto = 4;
	int *matriz;
	matriz = (int*)malloc(ancho*alto * sizeof(int));
	for (int i = 0; i < ancho*alto; i++) {
		matriz[i] = 0;
	}

	
	while (partida_enCurso) 
	{
		char movimiento = 'p';
		printf("Tablero:\n");
		generateSeeds(matriz, 5);
		showMatriz(matriz, 4);
		printf("Hacia donde quieres mover?(w/a/s/d): ");
		cin >> movimiento;
		switch (movimiento)
		{
		case 'w':
			cudaStatus = move_up(matriz,ancho,alto);
		case 'a':
		case 's':
			cudaStatus = move_down(matriz, ancho, alto);
		case 'd':

		default:
			break;
		}
		//system("CLS");
	}

	

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	
    return 0;
}

// Metodo que SOLO muestra matrices cuadradas
void showMatriz(int matriz[], int anchura)
{
	for (int i = 0; i < (anchura*anchura); i++) {
		printf("%d	", matriz[i]);
		if ((i + 1) % anchura == 0) {
			printf("\n");
		}
	}
}

int * generateMatriz()
{
	const int ancho = 4;
	const int alto = 4;
	int c[ancho*alto] = { 0 };
	return c;
}

void generateSeeds(int matriz[],int cantidad)
{
	for (int i = 0; i < cantidad; i++)
	{
		int r = rand() % 16;
		while (matriz[r] != 0) {
			r = rand() % 16;
		}

		int opcion = rand() % 100;
		if (opcion <= 50) {
			matriz[r] = 2;
		}
		else if (opcion <= 80 && opcion > 50) {
			matriz[r] = 4;
		}
		else {
			matriz[r] = 8;
		}
	}
	
	
}

bool checkFull(int matriz[],int tamano) 
{
	for (int i = 0; i < tamano; i++) 
	{
		if (matriz[i] == 0)
		{
			return false;
		}
	}
	return true;
}