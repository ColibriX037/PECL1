#include ""
#include <time.h>
#include <stdlib.h>
#include <stdio.h>


int main()
{

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

// Metodo que SOLO muestra matrices cuadradas
void showMatriz(int matriz[], int anchura)
{
	for (int i = 0; i < (anchura*anchura); i++) {
		printf("%d	", matriz[i]);
		if ((i + 1) % anchura == 0) {
			printf("\n");
		}
	}
}

int * generateMatriz()
{
	const int ancho = 4;
	const int alto = 4;
	int c[ancho*alto] = { 0 };
	return c;
}

void generateSeeds(int matriz[])
{
	srand(time(NULL));

	int r = rand()%16;
	while (matriz[r] == 0) {
		r = rand() % 16;
	}

	int opcion = rand() % 100;
	if (opcion <= 60) {
		matriz[r] = 2;
	}
	else {
		matriz[r] = 4;
	}
	/////////////////////////////
	int j = rand() % 16;
	while (matriz[j] == 0) {
		j = rand() % 16;
	}
	
	opcion = rand() % 100;
	if (opcion <= 60) {
		matriz[r] = 2;
	}
	else {
		matriz[r] = 4;
	}
}



bool checkFull(int matriz[],int tamano) 
{
	for (int i = 0; i < tamano; i++) 
	{
		if (matriz[i] == 0)
		{
			return false;
		}
	}
	return true;
}