#include "hip/hip_runtime.h"

#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

using namespace std;

void showMatriz(int matriz[], int anchura);
int * generateMatriz();
void generateSeeds(int matriz[]);
bool checkFull(int matriz[], int tamano);

hipError_t cudaStatus;
bool partida_enCurso = true;

__global__ void mov_up(int * matriz[]) {

}

hipError_t move_up(int * matriz[]) {

}

__global__ void mov_down(int * matriz[]) {

}

hipError_t move_down(int * matriz[]) {

}

__global__ void mov_left(int * matriz[]) {

}

hipError_t move_left(int * matriz[]) {

}

__global__ void mov_right(int * matriz[]) {

}

hipError_t move_right(int * matriz[]) {

}

int main()
{
	const int ancho = 4;
	const int alto = 4;
	int matriz[ancho*alto] = { 0 };

	while (partida_enCurso) 
	{
		char movimiento = 'p';
		printf("Tablero:\n");
		showMatriz(matriz, 4);
		printf("�Hacia donde quieres mover?(w/a/s/d): ");
		cin >> movimiento;
		switch (movimiento)
		{
		case 'w':
		case 'a':
		case 's':
		case 'd':

		default:
			break;
		}
		system("CLS");
	}

	

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Metodo que SOLO muestra matrices cuadradas
void showMatriz(int matriz[], int anchura)
{
	for (int i = 0; i < (anchura*anchura); i++) {
		printf("%d	", matriz[i]);
		if ((i + 1) % anchura == 0) {
			printf("\n");
		}
	}
}

int * generateMatriz()
{
	const int ancho = 4;
	const int alto = 4;
	int c[ancho*alto] = { 0 };
	return c;
}

void generateSeeds(int matriz[])
{
	srand(time(NULL));

	int r = rand()%16;
	while (matriz[r] == 0) {
		r = rand() % 16;
	}

	int opcion = rand() % 100;
	if (opcion <= 60) {
		matriz[r] = 2;
	}
	else {
		matriz[r] = 4;
	}
	/////////////////////////////
	int j = rand() % 16;
	while (matriz[j] == 0) {
		j = rand() % 16;
	}
	
	opcion = rand() % 100;
	if (opcion <= 60) {
		matriz[r] = 2;
	}
	else {
		matriz[r] = 4;
	}
}

bool checkFull(int matriz[],int tamano) 
{
	for (int i = 0; i < tamano; i++) 
	{
		if (matriz[i] == 0)
		{
			return false;
		}
	}
	return true;
}