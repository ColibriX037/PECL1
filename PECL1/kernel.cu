#include "hip/hip_runtime.h"

#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

using namespace std;

void showMatriz(int matriz[], int anchura);
int * generateMatriz();
void generateSeeds(int matriz[]);
bool checkFull(int matriz[], int tamano);

hipError_t cudaStatus;
bool partida_enCurso = true;

__global__ void mov_upK(int *matriz,int *resultado,int anchura, int altura) {
	int posicion = blockIdx.x*anchura + threadIdx.x;

	int superior = posicion - anchura;

	if (posicion >= anchura)
	{
			if (matriz[posicion] == matriz[superior])
			{
				resultado[superior] = matriz[superior] * 2;
			}
			else if (matriz[superior] == 0)
			{
				resultado[superior] = matriz[posicion];
			}
			/*
			else if (superior - anchura < 0) {
				if (matriz[superior] == matriz[superior - anchura]) {
					_sleep(10);
					matriz[superior] = matriz[posicion];
					matriz[posicion] = 0;
				}
			}*/
	}
}

hipError_t move_up(int *matriz) {
	hipError_t cudaStatus;
	
	const int ancho = 4;
	const int alto = 4;

	int *dev_m;
	int *dev_resultado;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_resultado, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto*sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en synchronize de mov_upK");
		goto Error;
	}

	mov_upK <<< 1,ancho*alto  >>> (dev_m,dev_resultado,ancho,alto);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en synchronize de mov_upK");
		goto Error;
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_resultado, ancho*alto*sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__global__ void mov_downK(int * matriz[]) {

}

hipError_t move_down(int * matriz[]) {
	hipError_t cudaStatus;
	return cudaStatus;
}

__global__ void mov_leftK(int * matriz[]) {

}

hipError_t move_left(int * matriz[]) {
	hipError_t cudaStatus;
	return cudaStatus;
}

__global__ void mov_rightK(int * matriz[]) {

}

hipError_t move_right(int * matriz[]) {
	hipError_t cudaStatus;
	return cudaStatus;
}


int main()
{
	hipError_t cudaStatus;

	int ancho = 4;
	int alto = 4;
	int *matriz;
	matriz = (int*)malloc(ancho*alto * sizeof(int));
	for (int i = 0; i < ancho*alto; i++) {
		matriz[i] = 0;
	}

	while (partida_enCurso) 
	{
		char movimiento = 'p';
		printf("Tablero:\n");
		generateSeeds(matriz);
		showMatriz(matriz, 4);
		printf("�Hacia donde quieres mover?(w/a/s/d): ");
		cin >> movimiento;
		switch (movimiento)
		{
		case 'w':
			cudaStatus = move_up(matriz);
		case 'a':
		case 's':
		case 'd':

		default:
			break;
		}
		//system("CLS");
	}

	

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Metodo que SOLO muestra matrices cuadradas
void showMatriz(int matriz[], int anchura)
{
	for (int i = 0; i < (anchura*anchura); i++) {
		printf("%d	", matriz[i]);
		if ((i + 1) % anchura == 0) {
			printf("\n");
		}
	}
}

int * generateMatriz()
{
	const int ancho = 4;
	const int alto = 4;
	int c[ancho*alto] = { 0 };
	return c;
}

void generateSeeds(int matriz[])
{
	srand(time(NULL));

	int r = rand()%16;
	while (matriz[r] != 0) {
		r = rand() % 16;
	}

	int opcion = rand() % 100;
	if (opcion <= 50) {
		matriz[r] = 2;
	}
	else if (opcion<=80 && opcion>50) {
		matriz[r] = 4;
	}
	else {
		matriz[r] = 8;
	}
	/////////////////////////////
	int j = rand() % 16;
	while (matriz[j] != 0 || j==r) {
		j = rand() % 16;
	}
	
	opcion = rand() % 100;
	if (opcion <= 60) {
		matriz[j] = 2;
	}
	else if (opcion <= 80 && opcion > 50) {
		matriz[j] = 4;
	}
	else {
		matriz[j] = 8;
	}
}

bool checkFull(int matriz[],int tamano) 
{
	for (int i = 0; i < tamano; i++) 
	{
		if (matriz[i] == 0)
		{
			return false;
		}
	}
	return true;
}