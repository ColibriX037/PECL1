#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

using namespace std;

void showMatriz(int matriz[], int anchura, int altura);
void generateSeeds(int matriz[],int ancho, int alto ,int cantidad,char modo);
void gestionSemillas(int *matriz, int ancho,int numeroSemillas, int alto, char modo);
bool checkFull(int matriz[], int tamano);
bool checkMove(int matriz[], int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();

hipError_t cudaStatus;
bool partida_enCurso = true;

__global__ void mov_upK(int *matriz, int anchura, int altura) {

	int x = threadIdx.x;

	int *vector = (int*)malloc(sizeof(int)*altura);
	for (int i = 0; i < altura; i++)
	{
		vector[i] = 0;
	}

	int *aux = (int*)malloc(sizeof(int)*altura);
	for (int i = 0; i < altura; i++)
	{
		aux[i] = 0;
	}


	int posicion_Vector = 0;
	for (int i = 0; i < altura; i++)
	{
		if (matriz[i*anchura + x] != 0) {
			vector[posicion_Vector] = matriz[i*anchura + x];
			posicion_Vector++;
		}
	}

	int posicion_aux = 0;
	for (int j = 0; j < altura; j++)
	{
		if (vector[j] == vector[j + 1])
		{
			aux[posicion_aux] = vector[j] * 2;
			j++;
		}
		else
		{
			aux[posicion_aux] = vector[j];
		}

		posicion_aux++;
	}

	for (int k = 0; k < altura; k++)
	{
		if (!aux[k])
		{
			aux[k] = 0;
		}
	}

	for (int i = 0; i < altura; i++)
	{
		matriz[i*anchura + x] = aux[i];
	}
}


hipError_t move_up(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto *sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	mov_upK <<< 1,ancho  >>> (dev_m,ancho,alto);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto *sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__global__ void mov_downK(int *matriz, int anchura, int altura) {
	int x = threadIdx.x;

	int *vector = (int*)malloc(sizeof(int)*altura);
	for (int i = 0; i < altura; i++)
	{
		vector[i] = 0;
	}

	int *aux = (int*)malloc(sizeof(int)*altura);
	for (int i = 0; i < altura; i++)
	{
		aux[i] = 0;
	}

	int posicion_Vector = 0;
	for (int i = altura - 1; i >= 0; i--)
	{
		if (matriz[i*anchura + x] != 0) {
			vector[posicion_Vector] = matriz[i*anchura + x];
			posicion_Vector++;
		}
	}

	int posicion_aux = 0;
	for (int j = 0; j < altura; j++)
	{
		if (vector[j] == vector[j + 1])
		{
			aux[posicion_aux] = vector[j] * 2;
			j++;
		}
		else
		{
			aux[posicion_aux] = vector[j];
		}
		posicion_aux++;
	}

	for (int k = 0; k < altura; k++)
	{
		if (!aux[k])
		{
			aux[k] = 0;
		}
	}
			
	for (int i = 0; i < altura; i++)
	{
		matriz[(altura-1-i)*anchura + x] = aux[i];
	}
}

hipError_t move_down(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	mov_downK << < 1, ancho >> > (dev_m, ancho, alto);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__global__ void mov_leftK(int *matriz, int anchura, int altura) {

	int x = threadIdx.x;

	int *vector = (int*)malloc(sizeof(int)*anchura);
	for (int i = 0; i < anchura; i++)
	{
		vector[i] = 0;
	}

	int *aux = (int*)malloc(sizeof(int)*anchura);
	for (int i = 0; i < anchura; i++)
	{
		aux[i] = 0;
	}


	int posicion_Vector = 0;
	for (int i = 0; i < anchura; i++)
	{
		if (matriz[x*anchura + i] != 0) {
			vector[posicion_Vector] = matriz[x*anchura + i];
			posicion_Vector++;
		}
	}

	int posicion_aux = 0;
	for (int j = 0; j < anchura; j++)
	{
		if (vector[j] == vector[j + 1])
		{
			aux[posicion_aux] = vector[j] * 2;
			j++;
		}
		else
		{
			aux[posicion_aux] = vector[j];
		}

		posicion_aux++;
	}

	for (int k = 0; k < anchura; k++)
	{
		if (!aux[k])
		{
			aux[k] = 0;
		}
	}

	for (int i = 0; i < anchura; i++)
	{
		matriz[x*anchura + i] = aux[i];
	}
}

hipError_t move_left(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	mov_leftK << < 1, ancho >> > (dev_m, ancho, alto);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__global__ void mov_rightK(int *matriz, int anchura, int altura) {

	int x = threadIdx.x;

	int *vector = (int*)malloc(sizeof(int)*anchura);
	for (int i = 0; i < anchura; i++)
	{
		vector[i] = 0;
	}

	int *aux = (int*)malloc(sizeof(int)*anchura);
	for (int i = 0; i < anchura; i++)
	{
		aux[i] = 0;
	}


	int posicion_Vector = 0;
	for (int i = anchura-1; i >=0; i--)
	{
		if (matriz[x*anchura + i] != 0) {
			vector[posicion_Vector] = matriz[x*anchura + i];
			posicion_Vector++;
		}
	}

	int posicion_aux = 0;
	for (int j = 0; j < anchura; j++)
	{
		if (vector[j] == vector[j + 1])
		{
			aux[posicion_aux] = vector[j] * 2;
			j++;
		}
		else
		{
			aux[posicion_aux] = vector[j];
		}

		posicion_aux++;
	}

	for (int k = 0; k < anchura; k++)
	{
		if (!aux[k])
		{
			aux[k] = 0;
		}
	}

	for (int i = 0; i < anchura; i++)
	{
		matriz[(x*anchura) + (anchura-1-i)] = aux[i];
	}
}

hipError_t move_right(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	mov_rightK << < 1, ancho >> > (dev_m, ancho, alto);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}


int main()
{
	hipError_t cudaStatus;
	srand(time(NULL));

	int ancho;
	int alto;
	int numSemillas = 0;
	int vidas = 5;
	char modo;
	char cargado;
	int *datos;
	int *matriz;

	printf("Desea comprobar si hay partidas guardadas?(y/n): ");
	cin >> cargado;
	if (cargado == 'y') 
	{
		datos = cargar();

		vidas = datos[0];
		alto = datos[1];
		ancho = datos[2];

		int dificultad = datos[3];

		if(dificultad == 0)
		{
			modo = 'B';
			numSemillas = 15;
		}
		else
		{
			modo = 'A';
			numSemillas = 8;
		}

		matriz = (int*)malloc(ancho*alto * sizeof(int));

		for (int i = 0; i < alto*ancho; i++)
		{
			matriz[i] = datos[4 + i];
		}
	}
	else
	{
		printf("Indique el ancho de la matriz: ");
		cin >> ancho;
		printf("Indique el alto de la matriz: ");
		cin >> alto;
		printf("Indique la dificultad del juego (B->Bajo / A->Alto): ");
		cin >> modo;
		switch (modo)
		{
		case 'B':
			numSemillas = 15;
			break;
		case 'A':
			numSemillas = 8;
			break;
		default:
			break;
		}



		matriz = (int*)malloc(ancho*alto * sizeof(int));
		for (int i = 0; i < ancho*alto; i++) {
			matriz[i] = 0;
		}
	}
	

	while (!checkFull(matriz,ancho*alto) || checkMove(matriz,ancho,alto)) 
	{
		system("CLS");

		gestionSemillas(matriz, ancho,numSemillas, alto, modo);

		char movimiento = 'p';
		printf("Vidas restantes: %d\n", vidas);
		printf("Tablero:\n");
		showMatriz(matriz, ancho,alto);
		printf("Hacia donde quieres mover?(w/a/s/d) Para guardar teclee g: ");
		cin >> movimiento;
		switch (movimiento)
		{
		case 'w':
			cudaStatus = move_up(matriz,ancho,alto);
			break;
		case 'a':
			cudaStatus = move_left(matriz, ancho, alto);
			break;
		case 's':
			cudaStatus = move_down(matriz, ancho, alto);
			break;
		case 'd':
			cudaStatus = move_right(matriz, ancho, alto);
			break;
		case 'g':
			guardar(vidas,matriz,alto,ancho,modo);
			printf("Partida guardada, hasta pronto!");
			return 0;
			break;
		default:
			break;
		}
		
		
		if (!(!checkFull(matriz, ancho*alto) || checkMove(matriz, ancho, alto)) && vidas > 0)
		{
			for (int i = 0; i < ancho*alto; i++) {
				matriz[i] = 0;
			}
			vidas--;
		}
	
	}

	

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	
    return 0;
}

// Metodo que SOLO muestra matrices cuadradas
void showMatriz(int matriz[], int anchura , int altura)
{
	for (int i = 0; i < altura; i++)
	{
		for (int j = 0; j < anchura; j++)
		{
			printf("%d\t", matriz[i*anchura + j]);
		}
		printf("\n");
	}
}

void generateSeeds(int matriz[], int ancho, int alto,int cantidad, char modo)
{
	int total = ancho * alto;
	int num;

	if (modo == 'B') 
	{
		
		for (int i = 0; i < cantidad; i++)
		{
			int r = rand() % total;
			while (matriz[r] != 0) {
				r = rand() % total;
			}

			int opcion = rand() % 100;
			if (opcion <= 50) {
				matriz[r] = 2;
			}
			else if (opcion <= 80 && opcion > 50) {
				matriz[r] = 4;
			}
			else {
				matriz[r] = 8;
			}
		}
	}
	else if (modo == 'A')
	{
		for (int i = 0; i < cantidad; i++)
		{
			int r = rand() % total;
			while (matriz[r] != 0) {
				r = rand() % total;
			}

			int opcion = rand() % 100;
			if (opcion <= 60) {
				matriz[r] = 2;
			}
			else {
				matriz[r] = 4;
			}

		}
	}
	
	
	
}

bool checkMove(int matriz[], int ancho, int alto)
{
    int contador = 0;
    int paso = 0;
    for (int i = 0; i < alto-1; i++)
    {
        for ( int j = 0; j < ancho - 1; j++)
        {
            if (matriz[paso] == matriz[paso + ancho] && matriz[paso + ancho]==0)
                return true;
            if (matriz[paso] == matriz[paso + 1] && matriz[paso + 1] == 0)
                return true;
            paso++;
        }
        paso = paso + 2;

    }

    paso = paso + ancho-1;

    for (int k = 0; k < alto - 1; k++) 
    {
        if (matriz[paso] == matriz[paso + ancho] && matriz[paso + ancho] == 0)
            return true;
    }

    paso = ancho*alto-2;

    for (int l = 0; l < ancho-2; l++) 
    {
        if (matriz[paso] == matriz[paso + 1] && matriz[paso + ancho] == 0)
            return true;
        paso--;
    }

	return false;

}

bool checkFull(int matriz[],int tamano) 
{
	for (int i = 0; i < tamano; i++) 
	{
		if (matriz[i] == 0)
		{
			return false;
		}
	}
	return true;
}

void gestionSemillas(int *matriz, int ancho,int numeroSemillas, int alto, char modo)
{
	if (!checkFull(matriz, ancho*alto))
	{
		int n = 0;
		for (int i = 0; i < ancho*alto; i++)
		{
			if (matriz[i] == 0)
				n++;
		}
		if (modo == 'B')
		{
			if (n < 15)
			{
				generateSeeds(matriz, ancho, alto, n, modo);
			}
			else {
				generateSeeds(matriz, ancho, alto, numeroSemillas, modo);
			}
			
		}
		
	}
}

void guardar(int vidas, int *matriz, int altura, int anchura, char dificultad) {

	ofstream archivo;
	int dif;

	archivo.open("2048_savedata.txt", ios::out); //Creamos o reemplazamos el archivo

	//Si no se puede guardar ERROR
	if (archivo.fail())
	{
		cout << "Error al guardar la partida.\n";
		exit(1);
	}

	if (dificultad == 'B')
	{
		dif = 0;
	}
	else
	{
		dif = 1;
	}

	archivo << vidas << endl; //Guardamos las vidas
	archivo << altura << endl; //Guardamos las altura
	archivo << anchura << endl; //Guardamos las anchura
	archivo << dif << endl; //Guardamos la dificultad

	//Guardamos la matriz
	for (int i = 0; i < (altura*anchura); i++)
	{
		archivo << matriz[i] << " ";
	}
	cout << "\nPartida guardada con exito." << endl;

	archivo.close(); //Cerramos el archivo
}

int* cargar() {

	ifstream archivo;
	int i = 4, vidas, altura, anchura, dif;
	int *partida;

	archivo.open("2048_savedata.txt", ios::in); //Abrimos el archivo en modo lectura

	//Si no se puede cargar ERROR
	if (archivo.fail())
	{
		cout << "Error al abrir la partida guardada. El fichero no existe o est� corrupto\n";
		exit(1);
	}

	archivo >> vidas;
	archivo >> altura;
	archivo >> anchura;
	archivo >> dif;

	partida = (int*)malloc(altura * anchura * sizeof(int)); //Reservamos memoria para los datos de la partida

	partida[0] = vidas; //Guardamos vidas
	partida[1] = altura; //Guardamos altura
	partida[2] = anchura; //Guardamos anchura
	partida[3] = dif; //Guardamos la dificultad

	//Guardamos la matriz
	while (!archivo.eof()) { //Mientras no sea el final del archivo
		archivo >> partida[i];
		i++;
	}

	archivo.close(); //Cerramos el archivo

	return partida;
}